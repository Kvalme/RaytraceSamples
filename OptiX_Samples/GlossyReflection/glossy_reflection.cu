#include "hip/hip_runtime.h"
/**********************************************************************
Copyright (c) 2016 Advanced Micro Devices, Inc. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
********************************************************************/

#include <optix.h>
#include <optixu/optixu_matrix.h>
#include <utils.h>
#include <sampler.h>
#include "microfacetggx.h"

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned int, primary_ray_type, , );
rtDeclareVariable(unsigned int, indirect_ray_type, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(unsigned int, frame_no,,);

rtDeclareVariable(CameraParams, camera_params, , );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(float3, texcoord,         attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, color, attribute color, );
rtDeclareVariable(float3, pos, attribute pos, );


rtDeclareVariable(float3, back_hit_point,   attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,  attribute front_hit_point, ); 


rtBuffer<float4, 2>   output_buffer;

#define DENOM_EPS 1e-8f
#define ROUGHNESS 0.01f

RT_PROGRAM void GenerateCameraRays()
{
    // Get hold of the pixel
    float2 pixelPos = make_float2(launch_index.x, launch_index.y);
    // Convert to world space position
    float2 ndc = 2.0f * (pixelPos + 0.5f) * make_float2(camera_params.screen_dims.z, camera_params.screen_dims.w) - 1.0f;

    float2 ndc2 = ndc * make_float2(1.0f, -1.0f);

    float4 homogeneous = camera_params.view_proj_inv * make_float4(ndc2.x, ndc2.y, 0.0f, 1.0f);
    homogeneous.x /= homogeneous.w; // projection divide
    homogeneous.y /= homogeneous.w; // projection divide
    homogeneous.z /= homogeneous.w; // projection divide
    homogeneous.w /= homogeneous.w; // projection divide

    float3 eye = make_float3(camera_params.eye.x, camera_params.eye.y, camera_params.eye.z);

    // Create the camera ray
    optix::Ray ray(eye, normalize(make_float3(homogeneous.x, homogeneous.y, homogeneous.z) - eye), primary_ray_type, scene_epsilon);

    PerRayData_radiance prd;
    prd.importance = 1.f;

    rtTrace(top_object, ray, prd);

    // Write the ray out to memory
    output_buffer[launch_index] += prd.result;
}

RT_PROGRAM void MissHitPrimary()
{
    prd_radiance.result = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
}

RT_PROGRAM void ShadePrimaryRays()
{
    float pdf;
    float3 bxdf;
    float3 wo;
    float3 wi = -normalize(ray.direction);

    Sampler sampler;
    Sampler_Init(&sampler, launch_index.x + launch_index.y * 1920 + frame_no);;

    float2 sample = Sampler_Sample2D(&sampler);

    bxdf = MicrofacetGGX_Sample(ROUGHNESS, color, wi, sample, shading_normal, &wo, &pdf);

    optix::Ray indirect_ray(pos + shading_normal * 0.001f, normalize(wo), indirect_ray_type, scene_epsilon);

    prd_radiance.result = make_float4(bxdf / pdf, 1.0f);
    
    PerRayData_radiance prd;
    rtTrace(top_object, indirect_ray, prd);

    prd_radiance.result = prd.result;
}

RT_PROGRAM void ClosestHitIndirectRay()
{
    float pdf;
    float3 bxdf;
    float3 wo;
    float3 wi = -normalize(ray.direction);

    Sampler sampler;
    Sampler_Init(&sampler, launch_index.x + launch_index.y * 1920 + frame_no);;

    float2 sample = Sampler_Sample2D(&sampler);


    bxdf = MicrofacetGGX_Sample(ROUGHNESS, color, wi, sample, shading_normal, &wo, &pdf);
    prd_radiance.result = make_float4(bxdf / pdf, 2.0f);
}

RT_PROGRAM void MissHitIndirectRay()
{
    prd_radiance.result = make_float4(1.f, 0.f, 0.f, 1.0f);
}

RT_PROGRAM void Exception()
{
  output_buffer[launch_index] = make_float4(1.0f, 0.0f, 1.0f, 0.0f);
}

/*
    GLOBAL Shape const* restrict shapes,
    GLOBAL Vertex const* restrict vertices,
    GLOBAL uint const* restrict indices,
    GLOBAL Ray* restrict output_rays,
    GLOBAL Ray* restrict input_rays,
    GLOBAL Intersection const* restrict isects,
    int intersection_count,
    GLOBAL float4* restrict output,
    GLOBAL float4* restrict color_buffer,
    volatile GLOBAL uint * restrict ao_rays_counter,
    GLOBAL uint const* restrict max_output_rays,
    int ao_rays_per_frame,
    int frame_no
)
{
    // Get hold of the pixel
    const int gid = get_global_id(0);
    const int pixel_id = input_rays[gid].padding.x;

    const Intersection hit = isects[gid];
    if (gid < intersection_count)
    {
        // Miss
        if (hit.shapeid == INVALID_IDX)
        {
            output[pixel_id] = (float4)(0.0f, 0.0f, 0.0f, 1.0f);
            return;
        }

        Shape shape = shapes[hit.shapeid];
        Vertex v0 = vertices[shape.base_vertex + indices[shape.first_index + 3 * hit.primid + 0]];
        Vertex v1 = vertices[shape.base_vertex + indices[shape.first_index + 3 * hit.primid + 1]];
        Vertex v2 = vertices[shape.base_vertex + indices[shape.first_index + 3 * hit.primid + 2]];

        float3 color = (1.0f - hit.uvwt.x - hit.uvwt.y) * v0.color + hit.uvwt.x * v1.color + hit.uvwt.y * v2.color;
        float3 pos = (1.0f - hit.uvwt.x - hit.uvwt.y) * v0.position + hit.uvwt.x * v1.position + hit.uvwt.y * v2.position;
        float3 normal = (1.0f - hit.uvwt.x - hit.uvwt.y) * v0.normal + hit.uvwt.x * v1.normal + hit.uvwt.y * v2.normal;

        // Write color to output buffer
        color_buffer[pixel_id] = (float4)(color, 1.0f);

        Sampler sampler;
        Sampler_Init(&sampler, gid + frame_no);

        //Get location index
        int ray_idx = atomic_add(ao_rays_counter, ao_rays_per_frame);
        if (ray_idx + ao_rays_per_frame < *max_output_rays)
        {
            for (int a = 0; a < ao_rays_per_frame; ++a)
            {
                float2 sample = Sampler_Sample2D(&sampler);
                float3 dir = Sample_MapToHemisphere(sample, normal, 0.f);

                Ray ray;
                ray.o = (float4)(pos + normal * 0.001f, 100000.f);
                ray.d = (float4)(dir, 0.f);
                ray.extra.x = 0xffffffff;
                ray.extra.y = 0xffffffff;
                ray.padding.x = pixel_id;
                output_rays[ray_idx + a] = ray;
            }
        }
    }
}

KERNEL
void ProcessAO(
    GLOBAL Ray* restrict input_rays,
    GLOBAL Intersection const* restrict isects,
    int intersection_count,
    GLOBAL float4* restrict color_buffer,
    GLOBAL float4* restrict output
)
{
    // Get hold of the pixel
    const int gid = get_global_id(0);
    const int pixel_id = input_rays[gid].padding.x;

    const Intersection hit = isects[gid];
    if (gid < intersection_count)
    {
        // Miss
        if (hit.shapeid == INVALID_IDX)
        {
            output[pixel_id] += color_buffer[pixel_id];
            return;
        }
        else
        {
            output[pixel_id] += (float4)(0.0f, 0.0f, 0.0f, 1.0f);
            return;
        }
    }
}

KERNEL
void Resolve(
    GLOBAL float4* restrict output,
    int frame_count
)
{
    // Get hold of the pixel
    const int gid = get_global_id(0);
    output[gid] /= output[gid].w > 0.9f ? output[gid].w : 1.0f;
}



*/
