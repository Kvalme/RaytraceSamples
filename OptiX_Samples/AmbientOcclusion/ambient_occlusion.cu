#include "hip/hip_runtime.h"
/**********************************************************************
Copyright (c) 2016 Advanced Micro Devices, Inc. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
********************************************************************/

#include <optix.h>
#include <optixu/optixu_matrix.h>
#include <utils.h>
#include <sampler.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned int, primary_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(unsigned int, ao_rays_per_frame, , );
rtDeclareVariable(unsigned int, frame_no,,);

rtDeclareVariable(CameraParams, camera_params, , );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(float3, texcoord,         attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, color, attribute color, );
rtDeclareVariable(float3, pos, attribute pos, );


rtDeclareVariable(float3, back_hit_point,   attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,  attribute front_hit_point, ); 


rtBuffer<float4, 2>   output_buffer;

RT_PROGRAM void GenerateCameraRays()
{
    // Get hold of the pixel
    float2 pixelPos = make_float2(launch_index.x, launch_index.y);
    // Convert to world space position
    float2 ndc = 2.0f * (pixelPos + 0.5f) * make_float2(camera_params.screen_dims.z, camera_params.screen_dims.w) - 1.0f;

    float2 ndc2 = ndc * make_float2(1.0f, -1.0f);

    float4 homogeneous = camera_params.view_proj_inv * make_float4(ndc2.x, ndc2.y, 0.0f, 1.0f);
    homogeneous.x /= homogeneous.w; // projection divide
    homogeneous.y /= homogeneous.w; // projection divide
    homogeneous.z /= homogeneous.w; // projection divide
    homogeneous.w /= homogeneous.w; // projection divide

    float3 eye = make_float3(camera_params.eye.x, camera_params.eye.y, camera_params.eye.z);

    // Create the camera ray
    optix::Ray ray(eye, normalize(make_float3(homogeneous.x, homogeneous.y, homogeneous.z) - eye), primary_ray_type, scene_epsilon);

    PerRayData_radiance prd;
    prd.importance = 1.f;

    rtTrace(top_object, ray, prd);

    // Write the ray out to memory
    output_buffer[launch_index] += prd.result;
}

RT_PROGRAM void MissHitPrimary()
{
    prd_radiance.result = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
}

RT_PROGRAM void ShadePrimaryRays()
{
    Sampler sampler;
    Sampler_Init(&sampler, launch_index.x +  launch_index.y * 1920 + frame_no);
    float3 c = make_float3(0.f, 0.f, 0.f);
    for (int a = 0; a < ao_rays_per_frame; ++a)
    {
        float2 sample = Sampler_Sample2D(&sampler);
        float3 dir = Sample_MapToHemisphere(sample, shading_normal, 0.f);

        optix::Ray ray(pos + shading_normal * 0.001f, dir, shadow_ray_type, scene_epsilon);
        rtTrace(top_object, ray, prd_radiance);
        c += color * prd_radiance.importance;
    }
    prd_radiance.result = make_float4(c.x, c.y, c.z, ao_rays_per_frame);
}

RT_PROGRAM void AnyHitShadowRay()
{
    prd_radiance.importance = 0.0f;
    rtTerminateRay();
}

RT_PROGRAM void MissHitShadowRay()
{
    prd_radiance.importance = 1.0f;
}

RT_PROGRAM void Exception()
{
  output_buffer[launch_index] = make_float4(1.0f, 0.0f, 1.0f, 0.0f);
}
