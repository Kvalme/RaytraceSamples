#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <scene.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float4, camera_eye, , );
rtDeclareVariable(float4, camera_near_far, , );
rtDeclareVariable(float4, camera_screen_dims, , );
rtDeclareVariable(float4, eye, , );

optix::Matrix4x4 view_proj_inv;


rtBuffer<float4, 2>   output_buffer;

rtDeclareVariable(CameraParams, )

/*


rtDeclareVariable(float3,                draw_color, , );

RT_PROGRAM void draw_solid_color()
{
  output_buffer[launch_index] = make_float4(draw_color, 0.f);
}
*/

rtDeclareVariable()

RT_PROGRAM void GenerateCameraRays()
{
    // Get hold of the pixel
    const int gid = get_global_id(0);

    float2 pixelPos = (float2)(gid % output_width, gid / output_width);

    // Convert to world space position
    float2 ndc = 2.0f * (pixelPos + 0.5f) * camera_params->screen_dims.zw - 1.0f;

    float4 homogeneous = matrix_mul_vector4(camera_params->view_proj_inv, (float4)(ndc * (float2)(1.0f, -1.0f), 0.0f, 1.0f));
    homogeneous.xyz /= homogeneous.w; // projection divide


                                      // Create the camera ray
    Ray ray;

    ray.d = (float4)(normalize(homogeneous.xyz - camera_params->eye.xyz), 0.0f);
    ray.o = camera_params->eye;
    ray.o.w = 100000.f;
    ray.extra.x = 0xffffffff;
    ray.extra.y = 0xffffffff;
    ray.padding.x = gid;

    // Write the ray out to memory
    rays[gid] = ray;
}


RT_PROGRAM void ShadePrimaryRays(
    GLOBAL Shape const* restrict shapes,
    GLOBAL Vertex const* restrict vertices,
    GLOBAL uint const* restrict indices,
    GLOBAL Ray* restrict output_rays,
    GLOBAL Ray* restrict input_rays,
    GLOBAL Intersection const* restrict isects,
    int intersection_count,
    GLOBAL float4* restrict output,
    GLOBAL float4* restrict color_buffer,
    volatile GLOBAL uint * restrict ao_rays_counter,
    GLOBAL uint const* restrict max_output_rays,
    int ao_rays_per_frame,
    int frame_no
)
{
    // Get hold of the pixel
    const int gid = get_global_id(0);
    const int pixel_id = input_rays[gid].padding.x;

    const Intersection hit = isects[gid];
    if (gid < intersection_count)
    {
        // Miss
        if (hit.shapeid == INVALID_IDX)
        {
            output[pixel_id] = (float4)(0.0f, 0.0f, 0.0f, 1.0f);
            return;
        }

        Shape shape = shapes[hit.shapeid];
        Vertex v0 = vertices[shape.base_vertex + indices[shape.first_index + 3 * hit.primid + 0]];
        Vertex v1 = vertices[shape.base_vertex + indices[shape.first_index + 3 * hit.primid + 1]];
        Vertex v2 = vertices[shape.base_vertex + indices[shape.first_index + 3 * hit.primid + 2]];

        float3 color = (1.0f - hit.uvwt.x - hit.uvwt.y) * v0.color + hit.uvwt.x * v1.color + hit.uvwt.y * v2.color;
        float3 pos = (1.0f - hit.uvwt.x - hit.uvwt.y) * v0.position + hit.uvwt.x * v1.position + hit.uvwt.y * v2.position;
        float3 normal = (1.0f - hit.uvwt.x - hit.uvwt.y) * v0.normal + hit.uvwt.x * v1.normal + hit.uvwt.y * v2.normal;

        // Write color to output buffer
        color_buffer[pixel_id] = (float4)(color, 1.0f);

        Sampler sampler;
        Sampler_Init(&sampler, gid + frame_no);

        //Get location index
        int ray_idx = atomic_add(ao_rays_counter, ao_rays_per_frame);
        if (ray_idx + ao_rays_per_frame < *max_output_rays)
        {
            for (int a = 0; a < ao_rays_per_frame; ++a)
            {
                float2 sample = Sampler_Sample2D(&sampler);
                float3 dir = Sample_MapToHemisphere(sample, normal, 0.f);

                Ray ray;
                ray.o = (float4)(pos + normal * 0.001f, 100000.f);
                ray.d = (float4)(dir, 0.f);
                ray.extra.x = 0xffffffff;
                ray.extra.y = 0xffffffff;
                ray.padding.x = pixel_id;
                output_rays[ray_idx + a] = ray;
            }
        }
    }
}

KERNEL
void ProcessAO(
    GLOBAL Ray* restrict input_rays,
    GLOBAL Intersection const* restrict isects,
    int intersection_count,
    GLOBAL float4* restrict color_buffer,
    GLOBAL float4* restrict output
)
{
    // Get hold of the pixel
    const int gid = get_global_id(0);
    const int pixel_id = input_rays[gid].padding.x;

    const Intersection hit = isects[gid];
    if (gid < intersection_count)
    {
        // Miss
        if (hit.shapeid == INVALID_IDX)
        {
            output[pixel_id] += color_buffer[pixel_id];
            return;
        }
        else
        {
            output[pixel_id] += (float4)(0.0f, 0.0f, 0.0f, 1.0f);
            return;
        }
    }
}

KERNEL
void Resolve(
    GLOBAL float4* restrict output,
    int frame_count
)
{
    // Get hold of the pixel
    const int gid = get_global_id(0);
    output[gid] /= output[gid].w > 0.9f ? output[gid].w : 1.0f;
}



